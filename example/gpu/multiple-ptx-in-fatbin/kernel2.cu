// kernel2.cu
#include <hip/hip_runtime.h>

extern __device__ int shared_counter;

__global__ void vectorMul(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
        atomicAdd(&shared_counter, 1);
    }
}
