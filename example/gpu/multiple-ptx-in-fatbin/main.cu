#include <hip/hip_runtime.h>
#include <stdio.h>

extern __global__ void vectorAdd(float *a, float *b, float *c, int n);
extern __global__ void vectorMul(float *a, float *b, float *c, int n);
extern __global__ void vectorSum(float *input, float *output, int n);

extern __device__ int shared_counter;
#define CHECK_CUDA(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

int main() {
    const int N = 1024;
    const int bytes = N * sizeof(float);
    
    float *h_a = (float*)malloc(bytes);
    float *h_b = (float*)malloc(bytes);
    float *h_c = (float*)malloc(bytes);
    float h_sum = 0.0f;
    
    for (int i = 0; i < N; i++) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }
    
    float *d_a, *d_b, *d_c, *d_sum;
    CHECK_CUDA(hipMalloc(&d_a, bytes));
    CHECK_CUDA(hipMalloc(&d_b, bytes));
    CHECK_CUDA(hipMalloc(&d_c, bytes));
    CHECK_CUDA(hipMalloc(&d_sum, sizeof(float)));
    
    CHECK_CUDA(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice));
    
    int counter_init = 0;
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(shared_counter), &counter_init, sizeof(int)));
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    
    // 启动 Kernel 1: 向量加法
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    
    // 拷贝结果回主机
    CHECK_CUDA(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));
    for (int i = 0; i < 5; i++) {
        printf("%.1f ", h_c[i]);
    }
    puts("\n\n");    
    vectorMul<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    
    CHECK_CUDA(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost));
    for (int i = 0; i < 5; i++) {
        printf("%.1f ", h_c[i]);
    }
    printf("\n\n");
    
    int smemSize = threadsPerBlock * sizeof(float);
    vectorSum<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_a, d_sum, N);
    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());
    
    CHECK_CUDA(hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost));
    printf("Sum: %.1f\n\n", h_sum);
    
    int final_counter;
    CHECK_CUDA(hipMemcpyFromSymbol(&final_counter, HIP_SYMBOL(shared_counter), sizeof(int)));
    printf("Shared counter: %d\n", final_counter);
    printf("(should be equal to N + N + blocksPerGrid = %d)\n", N + N + blocksPerGrid);
    
    // 清理
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_sum);
    
    return 0;
}
