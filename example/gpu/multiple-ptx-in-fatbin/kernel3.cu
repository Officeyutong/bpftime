// kernel3.cu
#include <hip/hip_runtime.h>

extern __device__ int shared_counter;

__global__ void vectorSum(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    sdata[tid] = (idx < n) ? input[idx] : 0.0f;
    __syncthreads();
    
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        atomicAdd(output, sdata[0]);
        atomicAdd(&shared_counter, 1);
    }
}
