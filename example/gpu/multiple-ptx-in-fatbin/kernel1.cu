// kernel1.cu
#include <hip/hip_runtime.h>

__device__ int shared_counter = 0;

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
        atomicAdd(&shared_counter, 1);
    }
}
